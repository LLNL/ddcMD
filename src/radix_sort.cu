#include "hip/hip_runtime.h"
#include "radix_sort.h"
/*
   CSci 360 Computer Architecture 3
   Hunter College of the City University of New York
   Prof. Stewart Weiss

   CUDA-based Parallel Radix Sort 

   For complete details and an article about other approaches, see
   http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

   This algorithm is due to
    Mark Harris
    NVIDIA Corporation

    Shubhabrata Sengupta
    University of California, Davis

    John D. Owens
    University of California, Davis 

   I have rewritten parts of it to make it clearer, and provided a proof
   of correctness for the partition step.

 */


// Forward declaration of partition_by_bit(), called by radix_sort()
__device__ void partition_by_bit(unsigned int *keys, int *values, unsigned int bit);


__global__ void radix_sort(unsigned int *keys, int *values);

#ifdef __cplusplus
extern "C" {
#endif

void runRadixSort(unsigned *keys, int *values, int n) {
        radix_sort<<<1, n>>>(keys, values);

    }
#ifdef __cplusplus
}
#endif

/*******************************************************************************
   RADIX_SORT()

   For each bit position from the least significant to the most significant,
   partition the elements so that all elements with a 0 in that bit position
   precede those with a 1 in that position, using a stable sort.
   When all bits have been so processed, the array is sorted.
   Reminder -- a sort is stable if the sort preserves the relative order of 
               equal elements.

   Because this is a device function (executed by each thread concurrently),
   after each partitioning step, the threads must execute __syncthreads() so
   that the array is guaranteed to be ready for the next step.
 *******************************************************************************/

__global__ void radix_sort(unsigned int *keys, int *values) {
    int bit;
    for (bit = 0; bit < 32; ++bit) {
        partition_by_bit(keys, values, bit);
        __syncthreads();
    }
}

/*******************************************************************************

   PLUS_SCAN()

   plus_scan(a[]), where a[] is an array of integers, replaces a[] by the prefix
   sums of the elements of a. The prefix sum of an element in an array (or more 
   generally, any sequence) is the sum of all elements up to and including that
   element. The sum operation can be replaced by any binary associative operator,
   such as multiplication.

   A thread with ID i that calls plus_scan(a) gets as its return value the new 
   element in a[i]. All threads together collectively replace the elements of
   a[].
   

   Example:
                   A =  3  1  7  0  4  1  6  3 

   Successive iterations yield
       offset = 1  A =  3  4  8  7  4  5  7  9
       offset = 2  A =  3  4 11 11 12 12 11 14
       offset = 4  A =  3  4 11 11 15 16 22 25

   When it is finished it will have taken log N steps and used N log N adds.
   (This means that it is not work-efficient, since the sequential algorithm
    uses N adds.)

 *******************************************************************************/
template<class T>
__device__ T plus_scan(T *x) {
    unsigned int i = threadIdx.x; // id of thread executing this instance
    unsigned int n = blockDim.x; // total number of threads in this block
    unsigned int offset; // distance between elements to be added

    for (offset = 1; offset < n; offset *= 2) {
        T t;

        if (i >= offset)
            t = x[i - offset];

        __syncthreads();

        if (i >= offset)
            x[i] = t + x[i]; // i.e., x[i] = x[i] + x[i-1]

        __syncthreads();
    }
    return x[i];
}

/*******************************************************************************
    partition_by_bit()

    This function is executed by every thread. Given an array of non-negative
    integer values, and a bit position, b, this partitions the array such that
    for all values[i], i = 0,...,n-1, the value of bit b in each element 
    values[k]  for k < i is <= the value of bit b in values[i], and if bit b in
    values[j] == bit b in values[i], and j < i, then after the partition, the 
    two elements will be in the same relative order (i.e., it is a stable sort).

    Each thread is responsible for repositioning a single element of the array.
 *******************************************************************************/

__device__ void partition_by_bit(unsigned int *keys, int *values, unsigned int bit) {
    unsigned int i = threadIdx.x;
    unsigned int size = blockDim.x;
    int x_i = values[i]; // value of integer at position i
    unsigned int key = keys[i];
    unsigned int p_i = (key >> bit) & 1; // value of bit at position bit

    // Replace values array so that values[i] is the value of bit bit in
    // element i.
    keys[i] = p_i;

    // Wait for all threads to finish this.
    __syncthreads();

    // Now the values array consists of 0's and 1's, such that values[i] = 0
    // if the bit at position bit in element i was 0 and 1 otherwise.

    // Compute number of True bits (1-bits) up to and including values[i], 
    // transforming values[] so that values[i] contains the sum of the 1-bits
    // from values[0] .. values[i]
    unsigned int T_before = plus_scan(keys);
    /*
        plus_scan(values) returns the total number of 1-bits for all j such that
        j <= i. This is assigned to T_before, the number of 1-bits before i 
        (includes i itself)
     */

    // The plus_scan() function does not return here until all threads have
    // reached the __syncthreads() call in the last iteration of its loop
    // Therefore, when it does return, we know that the entire array has had
    // the prefix sums computed, and that values[size-1] is the sum of all
    // elements in the array, which happens to be the number of 1-bits in 
    // the current bit position.
    unsigned int T_total = keys[size - 1];
    // T_total, after the scan, is the total number of 1-bits in the entire array.

    unsigned int F_total = size - T_total;
    /*    
        F_total is the total size of the array less the number of 1-bits and hence
        is the number of 0-bits.
     */
    __syncthreads();

    /*
        The value x_i must now be put back into the values array in the correct
        position. The array has to satisfy the condition that all values with a 0 in
        the current bit position must precede all those with a 1 in that position
        and it must be stable, meaning that if x_j and x_k both had the same bit 
        value before, and j < k, then x_j must precede x_k after sorting.

        Therefore, if x_i had a 1 in the current bit position before, it must now
        be in the position such that all x_j that had a 0 precede it, and all x_j
        that had a 1 in that bit and for which j < i, must precede it. Therefore
        if x_i had a 1, it must go into the index T_before-1 + F_total, which is the
        sum of the 0-bits and 1-bits that preceded it before (subtracting 1 since
        T_before includes x_i itself).

        If x_i has a 0 in the current bit position, then it has to be "slid" down
        in the array before all x_j such that x_j has a 1 in the current bit, but 
        no farther than that. Since there are T_before such j, it has to go to
        position i - T_before.  (There are T_before such j because x_i had a zero,
        so in the prefix sum, it does not contribute to the sum.)
     */
    if (p_i) {
        values[T_before - 1 + F_total] = x_i;
        keys[T_before - 1 + F_total] = key;
    } else {
        values[i - T_before] = x_i;
        keys[i - T_before] = key;
    }
    /*
       The interesting thing is that no two values will be placed in the same 
       position. I.e., this is a permutation of the array.

       Proof: Suppose that x_i and x_j both end up in index k. There are three
       cases: 
         Case 1. x_i and x_j have a 1 in the current bit position 
         Since F_total is the same for all threads, this implies that T_before must
         be the same for threads i and j. But this is not possible because one must 
         precede the other and therefore the one that precedes it must have smaller
         T_before.

         Case 2.  x_i and x_j both have a 0 in the current bit position. 
         Since they both are in k, we have 
             k = i - T_bef_i = j - T_Bef_j  or
             i - j = T_bef_i - T_bef_j
         Assume i > j without loss of generality.  This implies that the number of
         1-bits from position j+1 to position i-1 (since both x_j and x_i have 
         0-bits) is i-j. But that is impossible since there are only i-j-2 positions
         from j+1 to i-1.

         Case 3. x_i and x_j have different bit values. 
         Assume without loss of generality that x_j has the 0-bit and x_i, the 1-bit.
         T_before_j is the number of 1 bits in positions strictly less than j, 
         because there is a 0 in position j. The total number of positions less than
         j is j, since the array is 0-based. Therefore:

         j-T_before_j is the number of 0-bits in positions strictly less than j. 
         This must be strictly less than F_total, since x_j has a 0 in position j, 
         so there is at least one more 0 besides those below position j. Hence:

         (1)    F_total > j - T_before_j

         Turning to i, T_before_i is at least 1, since x_i has a 1 in its bit. So, 
         T_before_i - 1 is at least 0, and 

         (2)    T_before_i - 1 + F_total >= F_total. 

         Therefore, combining (1) and (2)

         (3)   T_before_i - 1 + F_total >= F_total  
                                        >  j - T_before_j

         But if x_i and x_j map to the same position, then 

         (4)   j - T_before_j  = T_before_i - 1 + F_total 
                               > j - T_before_j

         which is a contradiction since a number cannot be greater than itself!

         Therefore it is impossible for x_i and x_j to be placed in the same index
         if i != j.
     
     */

}

/*******************************************************************************
Example

Let the input sequence be

    11, 7, 8, 4

We will write these as 4-bit binary numbers:

    1011   0111   1000   0100

and apply the radix sort to the sequence. The for-loop will iterate from 
bit = 0 to bit = 3 since there are just 4 significant bits in these numbers.

When bit = 0, the array of values of the bits in position 0 is
    
       1      1      0      0
and the array of T_before values is

       1      2      2      2

T_total = F_total = 2 for this bit position.
The resulting values array becomes

    1000   0100   1011   0111 

Repeating this procedure when bit = 1, the array of values of bits in position 1

       0      0      1      1
and the T_before array is

       0      0      1      2

T_total = F_total = 2 for this bit position also
The resulting values array remains the same:

    1000   0100   1011   0111 

Repeating this procedure when bit = 2, the array of values of bits in position 2

       0      1      0      1
and the T_before array is

       0      1      1      2

T_total = F_total = 2 for this bit position also
The resulting values array remains the same:

    1000   1011   0100   0111 

Repeating this procedure when bit = 3, the array of values of bits in position 3

       1      1      0      0
and the T_before array is

       1      2      2      2

T_total = F_total = 2 for this bit position also
The resulting values array becomes:

    0100   0111   1000   1011 

which is a sorted array.




 *******************************************************************************/


